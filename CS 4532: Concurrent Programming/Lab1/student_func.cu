#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Green, and Blue is in it.
//The 'A' stands for Alpha and is used for transparency; it will be
//ignored in this homework.

//Each channel Red, Blue, Green, and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  int id_x = (blockIdx.y * blockDim.x) + threadIdx.x;
  
  uchar4 rgba_pixel = rgbaImage[id_x];
  
  float output_pixel = .299f*rgba_pixel.x + .587f*rgba_pixel.y + .114f*rgba_pixel.z; 
  
  greyImage[id_x] = output_pixel;
}

__global__
void rgba_to_greyscale_2(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  int id_x = (blockIdx.y * blockDim.x) + threadIdx.x;
  
  uchar4 rgba_pixel = rgbaImage[id_x];
  
  float ary[] ={rgba_pixel.x, rgba_pixel.y, rgba_pixel.z};
  
  float maxValue = 0, minValue = 0;
  
  for(int i=0;i<3;i++){
    if(ary[i] >= maxValue){
        maxValue = ary[i];
    }
    
    if(ary[i] <= minValue){
        minValue = ary[i];
    }
  }
  
  float output_pixel = (maxValue + minValue)/2; 
  
  greyImage[id_x] = output_pixel;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  const dim3 blockSize(numCols, 1, 1);  //TODO

  const dim3 gridSize( 1, numRows, 1);  //TODO

 // first kernel  
 //rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
 // Second kernel 
  rgba_to_greyscale_2<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

